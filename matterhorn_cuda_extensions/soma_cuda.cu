#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_utils.h"
#include "soma.h"

/*
LIF神经元的前向传播函数。
LIF神经元反应函数：
$$U_{i}^{l}(t)=H_{i}^{l}(t-1)+\frac{1}{τ_{m}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
Heaviside脉冲函数：
$$O_{i}^{l}(t)=u[U_{i}^{l}(t)]$$
硬重置：
$$H_{i}^{l}(t)=U_{i}^{l}(t)[1-O_{i}^{l}(t)]+u_{rest}O_{i}^{l}(t)$$
@params:
    o: at::Tensor 脉冲输出$O^{l}$
    u: at::Tensor 胞体电位$U^{l}$
    h: at::Tensor 胞体历史电位$H^{l}$
    x: at::Tensor 输入电位$X^{l}$
    time_steps: int 总时间步长
    shape: int 总空间大小
    u_init: at::Tensor 初始胞体电位$H^{l}(-1)$
    tau_m: at::Tensor 时间常数$τ_{m}$
    u_rest: float 静息电位$u_{rest}$
    u_threshold: float 阈电位$u_{th}$
*/
__global__ void fp_lif_heaviside_hard_cuda_kernel(float* o,
                                                  float* u,
                                                  float* h,
                                                  float* x,
                                                  int time_steps,
                                                  int shape,
                                                  float* u_init,
                                                  float* tau_m,
                                                  float u_rest,
                                                  float u_threshold) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * 1024 + j;
    if (idx >= shape) {
        return;
    }

    const float tau_m_val = tau_m[0];
    for (int t = 0; t < time_steps; t++) {
        const int cur_idx = t * shape + idx;
        float cur_x = x[cur_idx];
        float last_h = t ? h[cur_idx - shape] : u_init[idx];
        float cur_u = 0.0f;
        float cur_o = 0.0f;
        float cur_h = 0.0f;
        // $$U_{i}^{l}(t)=H_{i}^{l}(t-1)+\frac{1}{τ_{m}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
        cur_u = last_h + (1.0f / tau_m_val) * (0.0f - (last_h - u_rest) + cur_x);
        // $$O_{i}^{l}(t)=u[U_{i}^{l}(t)]$$
        cur_o = cur_u >= u_threshold ? 1.0f : 0.0f;
        // $$H_{i}^{l}(t)=U_{i}^{l}(t)[1-O_{i}^{l}(t)]+u_{rest}O_{i}^{l}(t)$$
        cur_h = cur_u * (1.0f - cur_o) + u_rest * cur_o;
        u[cur_idx] = cur_u;
        o[cur_idx] = cur_o;
        h[cur_idx] = cur_h;
    }
}

void fp_lif_heaviside_hard_cuda(float* o,
                                float* u,
                                float* h,
                                float* x,
                                int time_steps,
                                int shape,
                                float* u_init,
                                float* tau_m,
                                float u_rest,
                                float u_threshold) {
    hipError_t err;

    // i = blockIdx.y 为行
    // j = blockIdx.x * blockDim.x + threadIdx.x 为列
    dim3 blocks(DIVUP(DIVUP(shape, 1024), THREADS_PER_BLOCK), 1024);
    dim3 threads(THREADS_PER_BLOCK);

    // 调用CUDA核心开始计算
    fp_lif_heaviside_hard_cuda_kernel<<<blocks, threads, 0>>>(
        o, u, h, x, time_steps, shape, u_init, tau_m, u_rest, u_threshold);

    // 返回计算结果
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

/*
LIF神经元的反向传播函数。
LIF神经元反应函数的反向传播：
$$\frac{\partial U_{i}^{l}(t)}{\partial H_{i}^{l}(t-1)}=1-\frac{1}{τ_{m}}$$
$$\frac{\partial U_{i}^{l}(t)}{\partial X_{i}^{l}(t)}=\frac{1}{τ_{m}}$$
$$\frac{\partial U_{i}^{l}(t)}{\partial
τ_{m}}=-\frac{1}{τ_{m}^{2}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
矩形窗反向传播：
$$\frac{\partial O_{i}^{l}(t)}{\partial U_{i}^{l}(t)}=u'$$
硬重置反向传播：
$$\frac{\partial H_{i}^{l}(t)}{\partial U_{i}^{l}(t)}=1-O_{i}^{l}(t)$$
$$\frac{\partial H_{i}^{l}(t)}{\partial O_{i}^{l}(t)}=-U_{i}^{l}(t)+u_{rest}$$
@params:
    grad_o: at::Tensor 脉冲输出$O^{l}$的梯度
    grad_u: at::Tensor 胞体电位$U^{l}$的梯度
    grad_h: at::Tensor 胞体历史电位$H^{l}$的梯度
    grad_x: at::Tensor 输入电位$X^{l}$的梯度
    grad_u_init: at::Tensor 初始胞体电位$H^{l}(-1)$的梯度
    grad_tau_m: at::Tensor 时间常数$τ_{m}$的梯度
    time_steps: int 总时间步长
    shape: int 总空间大小
    o: at::Tensor 脉冲输出$O^{l}$
    u: at::Tensor 胞体电位$U^{l}$
    h: at::Tensor 胞体历史电位$H^{l}$
    x: at::Tensor 输入电位$X^{l}$
    u_init: at::Tensor 初始胞体电位$H^{l}(-1)$
    tau_m: at::Tensor 时间常数$τ_{m}$
    u_rest: float 静息电位$u_{rest}$
    u_threshold: float 阈电位$u_{th}$
*/
__global__ void bp_lif_rectangular_hard_cuda_kernel(float* grad_o,
                                                    float* grad_u,
                                                    float* grad_h,
                                                    float* grad_x,
                                                    float* grad_u_init,
                                                    float* grad_tau_m,
                                                    int time_steps,
                                                    int shape,
                                                    float* o,
                                                    float* u,
                                                    float* h,
                                                    float* x,
                                                    float* u_init,
                                                    float* tau_m,
                                                    float u_rest,
                                                    float u_threshold) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * 1024 + j;
    if (idx >= shape) {
        return;
    }

    const float tau_m_val = tau_m[0];
    float cur_grad_h = 0.0f;
    for (int t = time_steps - 1; t >= 0; t--) {
        const int cur_idx = t * shape + idx;
        float last_h = t ? h[cur_idx - shape] : u_init[idx];
        float cur_x = x[cur_idx];
        float cur_u = u[cur_idx];
        float cur_o = o[cur_idx];
        float cur_grad_o = grad_o[cur_idx];
        float cur_grad_u = 0.0f;
        float cur_grad_x = 0.0f;
        float cur_grad_tau_m = grad_tau_m[0];
        // $$\frac{\partial H_{i}^{l}(t)}{\partial
        // U_{i}^{l}(t)}=1-O_{i}^{l}(t)$$
        // $$\frac{\partial H_{i}^{l}(t)}{\partial
        // O_{i}^{l}(t)}=-U_{i}^{l}(t)+u_{rest}$$
        cur_grad_o += cur_grad_h * (1.0f - cur_o);
        cur_grad_u += cur_grad_h * (0.0f - cur_u + u_rest);
        // $$\frac{\partial O_{i}^{l}(t)}{\partial U_{i}^{l}(t)}=u'$$
        cur_grad_u += cur_grad_o * 0.5f * (((cur_u > (u_threshold - 1.0f)) && (cur_u < (u_threshold + 1.0f))) ? 1.0f : 0.0f);
        // $$\frac{\partial U_{i}^{l}(t)}{\partial
        // H_{i}^{l}(t-1)}=1-\frac{1}{τ_{m}}$$
        // $$\frac{\partial U_{i}^{l}(t)}{\partial
        // X_{i}^{l}(t)}=\frac{1}{τ_{m}}$$
        // $$\frac{\partial U_{i}^{l}(t)}{\partial
        // τ_{m}}=-\frac{1}{τ_{m}^{2}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
        cur_grad_x += cur_grad_u * (1.0f / tau_m_val);
        cur_grad_h = cur_grad_u * (1.0f - (1.0f / tau_m_val));
        cur_grad_tau_m += cur_grad_u * (0.0f - (1.0f / (tau_m_val * tau_m_val)) * (0.0f - (last_h - u_rest) + cur_x));
        grad_o[cur_idx] = cur_grad_o;
        grad_u[cur_idx] = cur_grad_u;
        grad_x[cur_idx] = cur_grad_x;
        if (t) {
            grad_h[cur_idx - shape] = cur_grad_h;
        } else {
            grad_u_init[idx] = cur_grad_h;
        }
        grad_tau_m[0] = cur_grad_tau_m;
    }
}

void bp_lif_rectangular_hard_cuda(float* grad_o,
                                  float* grad_u,
                                  float* grad_h,
                                  float* grad_x,
                                  float* grad_u_init,
                                  float* grad_tau_m,
                                  int time_steps,
                                  int shape,
                                  float* o,
                                  float* u,
                                  float* h,
                                  float* x,
                                  float* u_init,
                                  float* tau_m,
                                  float u_rest,
                                  float u_threshold) {
    hipError_t err;

    // i = blockIdx.y 为行
    // j = blockIdx.x * blockDim.x + threadIdx.x 为列
    dim3 blocks(DIVUP(DIVUP(shape, 1024), THREADS_PER_BLOCK), 1024);
    dim3 threads(THREADS_PER_BLOCK);

    // 调用CUDA核心开始计算
    bp_lif_rectangular_hard_cuda_kernel<<<blocks, threads, 0>>>(
        grad_o, grad_u, grad_h, grad_x, grad_u_init, grad_tau_m, time_steps,
        shape, o, u, h, x, u_init, tau_m, u_rest, u_threshold);

    // 返回计算结果
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}