#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_utils.h"
#include "soma.h"

/*
LIF神经元的前向传播函数。
LIF神经元反应函数：
$$U_{i}^{l}(t)=H_{i}^{l}(t-1)+\frac{1}{τ_{m}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
Heaviside脉冲函数：
$$O_{i}^{l}(t)=u[U_{i}^{l}(t)]$$
硬重置：
$$H_{i}^{l}(t)=U_{i}^{l}(t)[1-O_{i}^{l}(t)]+u_{rest}O_{i}^{l}(t)$$
@params:
    o: at::Tensor 脉冲输出$O^{l}$
    u: at::Tensor 胞体电位$U^{l}$
    h: at::Tensor 胞体历史电位$H^{l}$
    x: at::Tensor 输入电位$X^{l}$
    time_steps: int 总时间步长
    shape: int 总空间大小
    u_init: at::Tensor 初始胞体电位$H^{l}(-1)$
    tau_m: at::Tensor 时间常数$τ_{m}$
    u_rest: float 静息电位$u_{rest}$
    u_threshold: float 阈电位$u_{th}$
*/
__global__ void fp_lif_heaviside_hard_cuda_kernel(float* o,
                                                  float* u,
                                                  float* h,
                                                  float* x,
                                                  int time_steps,
                                                  int shape,
                                                  float* u_init,
                                                  float* tau_m,
                                                  float u_rest,
                                                  float u_threshold) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * 1024 + j;
    if (idx >= shape) {
        return;
    }

    const float tau_m_val = tau_m[0];
    for (int t = 0; t < time_steps; t++) {
        const int cur_idx = t * shape + idx;
        float last_h = t ? h[cur_idx - shape] : u_init[idx];
        
        u[cur_idx] = last_h + (1.0f / tau_m_val) * (0.0f - (last_h - u_rest) + x[cur_idx]);
        
        o[cur_idx] = u[cur_idx] >= u_threshold ? 1.0f : 0.0f;
        
        h[cur_idx] = u[cur_idx] * (1.0f - o[cur_idx]) + u_rest * o[cur_idx];
    }
}

void fp_lif_heaviside_hard_cuda(float* o,
                                float* u,
                                float* h,
                                float* x,
                                int time_steps,
                                int shape,
                                float* u_init,
                                float* tau_m,
                                float u_rest,
                                float u_threshold) {
    hipError_t err;

    // i = blockIdx.y 为行
    // j = blockIdx.x * blockDim.x + threadIdx.x 为列
    dim3 blocks(DIVUP(DIVUP(shape, 1024), THREADS_PER_BLOCK), 1024);
    dim3 threads(THREADS_PER_BLOCK);

    // 调用CUDA核心开始计算
    fp_lif_heaviside_hard_cuda_kernel<<<blocks, threads, 0>>>(
        o, u, h, x, time_steps, shape, u_init, tau_m, u_rest, u_threshold);

    // 返回计算结果
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

/*
LIF神经元的反向传播函数。
LIF神经元反应函数的反向传播：
$$\frac{\partial U_{i}^{l}(t)}{\partial H_{i}^{l}(t-1)}=1-\frac{1}{τ_{m}}$$
$$\frac{\partial U_{i}^{l}(t)}{\partial X_{i}^{l}(t)}=\frac{1}{τ_{m}}$$
$$\frac{\partial U_{i}^{l}(t)}{\partial
τ_{m}}=-\frac{1}{τ_{m}^{2}}[-[H_{i}^{l}(t-1)-u_{rest}]+X_{i}^{l}(t)]$$
矩形窗反向传播：
$$\frac{\partial O_{i}^{l}(t)}{\partial U_{i}^{l}(t)}=u'$$
硬重置反向传播：
$$\frac{\partial H_{i}^{l}(t)}{\partial U_{i}^{l}(t)}=1-O_{i}^{l}(t)$$
$$\frac{\partial H_{i}^{l}(t)}{\partial O_{i}^{l}(t)}=-U_{i}^{l}(t)+u_{rest}$$
@params:
    grad_o: at::Tensor 脉冲输出$O^{l}$的梯度
    grad_u: at::Tensor 胞体电位$U^{l}$的梯度
    grad_h: at::Tensor 胞体历史电位$H^{l}$的梯度
    grad_x: at::Tensor 输入电位$X^{l}$的梯度
    grad_u_init: at::Tensor 初始胞体电位$H^{l}(-1)$的梯度
    grad_tau_m: at::Tensor 时间常数$τ_{m}$的梯度
    time_steps: int 总时间步长
    shape: int 总空间大小
    o: at::Tensor 脉冲输出$O^{l}$
    u: at::Tensor 胞体电位$U^{l}$
    h: at::Tensor 胞体历史电位$H^{l}$
    x: at::Tensor 输入电位$X^{l}$
    u_init: at::Tensor 初始胞体电位$H^{l}(-1)$
    tau_m: at::Tensor 时间常数$τ_{m}$
    u_rest: float 静息电位$u_{rest}$
    u_threshold: float 阈电位$u_{th}$
*/
__global__ void bp_lif_rectangular_hard_cuda_kernel(float* grad_o,
                                                    float* grad_u,
                                                    float* grad_h,
                                                    float* grad_x,
                                                    float* grad_u_init,
                                                    float* grad_tau_m,
                                                    int time_steps,
                                                    int shape,
                                                    float* o,
                                                    float* u,
                                                    float* h,
                                                    float* x,
                                                    float* u_init,
                                                    float* tau_m,
                                                    float u_rest,
                                                    float u_threshold) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * 1024 + j;
    if (idx >= shape) {
        return;
    }

    const float tau_m_val = tau_m[0];
    float cur_grad_h = 0.0f;
    for (int t = time_steps - 1; t >= 0; t--) {
        const int cur_idx = t * shape + idx;
        float last_h = t ? h[cur_idx - shape] : u_init[idx];

        grad_u[cur_idx] += cur_grad_h * (1.0f - o[cur_idx]);
        grad_o[cur_idx] += cur_grad_h * (0.0f - u[cur_idx] + u_rest);

        grad_u[cur_idx] += grad_o[cur_idx] * 0.5f * (((u[cur_idx] > (u_threshold - 1.0f)) && (u[cur_idx] < (u_threshold + 1.0f))) ? 1.0f : 0.0f);
        
        grad_x[cur_idx] += grad_u[cur_idx] * (1.0f / tau_m_val);
        cur_grad_h = grad_u[cur_idx] * (1.0f - (1.0f / tau_m_val));
        grad_tau_m[0] += grad_u[cur_idx] * (0.0f - (1.0f / (tau_m_val * tau_m_val)) * (0.0f - (last_h - u_rest) + x[cur_idx]));
        if (t) {
            grad_h[cur_idx - shape] = cur_grad_h;
        } else {
            grad_u_init[idx] = cur_grad_h;
        }
    }
}

void bp_lif_rectangular_hard_cuda(float* grad_o,
                                  float* grad_u,
                                  float* grad_h,
                                  float* grad_x,
                                  float* grad_u_init,
                                  float* grad_tau_m,
                                  int time_steps,
                                  int shape,
                                  float* o,
                                  float* u,
                                  float* h,
                                  float* x,
                                  float* u_init,
                                  float* tau_m,
                                  float u_rest,
                                  float u_threshold) {
    hipError_t err;

    // i = blockIdx.y 为行
    // j = blockIdx.x * blockDim.x + threadIdx.x 为列
    dim3 blocks(DIVUP(DIVUP(shape, 1024), THREADS_PER_BLOCK), 1024);
    dim3 threads(THREADS_PER_BLOCK);

    // 调用CUDA核心开始计算
    bp_lif_rectangular_hard_cuda_kernel<<<blocks, threads, 0>>>(
        grad_o, grad_u, grad_h, grad_x, grad_u_init, grad_tau_m, time_steps,
        shape, o, u, h, x, u_init, tau_m, u_rest, u_threshold);

    // 返回计算结果
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}